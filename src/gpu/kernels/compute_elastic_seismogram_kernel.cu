#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                        Princeton University, USA
!                and CNRS / University of Marseille, France
!                 (there are currently many more authors!)
! (c) Princeton University and CNRS / University of Marseille, April 2014
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/



//fortran code snippet...
/*
  ! gets global number of that receiver
  irec = number_receiver_global(irec_local)

  ! gets local receiver interpolators
  ! (1-D Lagrange interpolators)
  hxir(:) = hxir_store(irec_local,:)
  hetar(:) = hetar_store(irec_local,:)
  hgammar(:) = hgammar_store(irec_local,:)
*/

/* ----------------------------------------------------------------------------------------------- */

// unused...
/*
__device__ double my_atomicAdd(double* address, double val) {

    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do{
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
*/

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_elastic_seismogram_kernel(int nrec_local,
                                                  realw* field,
                                                  int* d_ibool,
                                                  realw* hxir, realw* hgammar,
                                                  realw* seismograms,
                                                  realw* cosrot,
                                                  realw* sinrot,
                                                  int* ispec_selected_rec_loc,
                                                  int it,
                                                  int NSTEP)
{


  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;

  int J = (tx/NGLLX);
  int I = (tx-J*NGLLX);

  __shared__ realw sh_dxd[NGLL2_PADDED];
  __shared__ realw sh_dzd[NGLL2_PADDED];


  if (irec_local < nrec_local) {

    int ispec = ispec_selected_rec_loc[irec_local] - 1;

    sh_dxd[tx] = 0;
    sh_dzd[tx] = 0;

    if (tx < NGLL2) {
      realw hlagrange = hxir[irec_local + nrec_local*I] * hgammar[irec_local + nrec_local*J];
      int iglob = d_ibool[tx+NGLL2_PADDED*ispec] - 1;

      sh_dxd[tx] = hlagrange * field[0 + 2*iglob];
      sh_dzd[tx] = hlagrange * field[1 + 2*iglob];

      //debug
      //if (tx == 0) printf("thread %d %d %d - %f %f %f\n",ispec,iglob,irec_local,hlagrange,field[0 + 2*iglob],field[1 + 2*iglob]);
    }
    __syncthreads();

    // reduction
    for (unsigned int s=1; s<NGLL2_PADDED ; s *= 2) {
      if (tx % (2*s) == 0){ sh_dxd[tx] += sh_dxd[tx + s];sh_dzd[tx] += sh_dzd[tx + s];}
      __syncthreads();
    }

    if (tx == 0) {seismograms[irec_local*NSTEP + it]                    = cosrot[irec_local]*sh_dxd[0]  + sinrot[irec_local]*sh_dzd[0];}
    if (tx == 1) {seismograms[irec_local*NSTEP + it + nrec_local*NSTEP] = cosrot[irec_local]*sh_dzd[0]  - sinrot[irec_local]*sh_dxd[0];}

    /*
    // simple, single-thread reduction
    if (tx == 0) {
      // a loop in thread 0 is faster than atomic operations
      for(int s=1;s<NGLL2;s++) {
        sh_dxd[0] += sh_dxd[s];
        sh_dzd[0] += sh_dzd[s];
      }

      // rotate seismogram components
      seismograms[irec_local]            =    cosrot[irec_local]*sh_dxd[0] + sinrot[irec_local]*sh_dzd[0];
      seismograms[irec_local+nrec_local] =  - sinrot[irec_local]*sh_dxd[0] + cosrot[irec_local]*sh_dzd[0];
    }
    */
  }
}

