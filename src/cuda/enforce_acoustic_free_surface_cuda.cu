#include "hip/hip_runtime.h"
/*
!========================================================================
!
!                   S P E C F E M 2 D  Version 7 . 0
!                   --------------------------------
!
!     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                              CNRS, France
!                       and Princeton University, USA
!                 (there are currently many more authors!)
!                           (c) October 2017
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".
!
!========================================================================
*/

#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

/* KERNEL for enforce free surface */

/* ----------------------------------------------------------------------------------------------- */


__global__ void enforce_free_surface_cuda_kernel(realw_p potential_acoustic,
                                                 realw_p potential_dot_acoustic,
                                                 realw_p potential_dot_dot_acoustic,
                                                 const int num_free_surface_faces,
                                                 const int* free_surface_ispec,
                                                 const int* free_surface_ij,
                                                 const int* d_ibool,
                                                 const int* ispec_is_acoustic) {
  // gets spectral element face id
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  // for all faces on free surface
  if (iface < num_free_surface_faces) {

    int ispec = free_surface_ispec[iface]-1;

    // checks if element is in acoustic domain
    if (ispec_is_acoustic[ispec]) {

      // gets global point index
      int igll = threadIdx.x + threadIdx.y*blockDim.x;

      int i = free_surface_ij[INDEX3(NDIM,NGLLX,0,igll,iface)] - 1; // (1,igll,iface)
      int j = free_surface_ij[INDEX3(NDIM,NGLLX,1,igll,iface)] - 1;

      int iglob = d_ibool[INDEX3_PADDED(NGLLX,NGLLX,i,j,ispec)] - 1;

      // sets potentials to zero at free surface
      potential_acoustic[iglob] = 0.f;
      potential_dot_acoustic[iglob] = 0.f;
      potential_dot_dot_acoustic[iglob] = 0.f;
    }
  }
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(acoustic_enforce_free_surf_cuda,
              ACOUSTIC_ENFORCE_FREE_SURF_CUDA)(long* Mesh_pointer,int* compute_wavefield_1,int* compute_wavefield_2) {

  TRACE("acoustic_enforce_free_surf_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // does not absorb free surface, thus we enforce the potential to be zero at surface

  // checks if anything to do
  if (mp->num_free_surface_faces == 0) return;

  // block sizes
  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->num_free_surface_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(NGLLX,1,1);


  // sets potentials to zero at free surface
  if (*compute_wavefield_1) {
  enforce_free_surface_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_acoustic,
                                                                          mp->d_potential_dot_acoustic,
                                                                          mp->d_potential_dot_dot_acoustic,
                                                                          mp->num_free_surface_faces,
                                                                          mp->d_free_surface_ispec,
                                                                          mp->d_free_surface_ijk,
                                                                          mp->d_ibool,
                                                                          mp->d_ispec_is_acoustic);
  }
  // for backward/reconstructed potentials
  if (*compute_wavefield_2) {
    enforce_free_surface_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_acoustic,
                                                                            mp->d_b_potential_dot_acoustic,
                                                                            mp->d_b_potential_dot_dot_acoustic,
                                                                            mp->num_free_surface_faces,
                                                                            mp->d_free_surface_ispec,
                                                                            mp->d_free_surface_ijk,
                                                                            mp->d_ibool,
                                                                            mp->d_ispec_is_acoustic);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("enforce_free_surface_cuda");
#endif
}

